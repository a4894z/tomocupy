#include "hip/hip_runtime.h"
#include "cfunc.cuh"
#include "kernels.cuh"
#include <stdio.h>

hipError_t copy3DDeviceToArray(hipArray* dfa, real* df, hipExtent ext, hipStream_t stream)
{
	hipMemcpy3DParms param = { 0 };
	param.srcPtr   = make_hipPitchedPtr((void*)df, ext.width*sizeof(real), ext.width, ext.height);
	param.dstArray = dfa;
	param.kind = hipMemcpyDeviceToDevice;
	param.extent = ext;
	return hipMemcpy3DAsync(&param,stream);
}

cfunc::cfunc(int nproj, int nz, int n, int ntheta, int nrho):
nproj(nproj), nz(nz), n(n), ntheta(ntheta), nrho(nrho) {

    // Create FFT plans for Fourier Transform in log-polar coordinates
    long long ffts[] = {nrho,ntheta};
	long long idist = nrho*ntheta;long long odist = nrho*(ntheta/2+1);
    long long inembed[] = {nrho, ntheta};long long onembed[] = {nrho, ntheta/2+1};
    size_t workSize = 0;
    hipfftCreate(&plan_forward);
    hipfftXtMakePlanMany(plan_forward, 
        2, ffts, 
        inembed, 1, idist, CUDA_R, 
        onembed, 1, odist, CUDA_C, 
        nz, &workSize, CUDA_C);    
    hipfftCreate(&plan_inverse);
    hipfftXtMakePlanMany(plan_inverse, 
        2, ffts, 
        onembed, 1, odist, CUDA_C, 
        inembed, 1, idist, CUDA_R, 
        nz, &workSize, CUDA_R);
    
    // Allocate temporarily arrays 
    hipMalloc((void **)&fl, nz*ntheta*nrho*sizeof(real)); 
    hipMalloc((void **)&flc, nz*(ntheta/2+1)*nrho*sizeof(complex)); 
    hipMalloc((void **)&gtmp, nz*n*nproj*sizeof(real)); 
    
    // Init texture references
    // texture texfl
    hipChannelFormatDesc texf_desc;
    texf_desc = CUDA_CREATE_CHANNEL_DESC();     
    hipExtent volumeSize = make_hipExtent(ntheta,nrho,nz); 
	
	hipMalloc3DArray(&fla, &texf_desc,volumeSize,hipArrayLayered); 
	texfl.addressMode[0] = hipAddressModeWrap;
	texfl.addressMode[1] = hipAddressModeWrap;
	texfl.filterMode = hipFilterModeLinear;
	texfl.normalized  = true;
    hipBindTextureToArray(texfl, fla,texf_desc); 

    // texture texg
    texf_desc = CUDA_CREATE_CHANNEL_DESC();    
    volumeSize = make_hipExtent(n,nproj,nz); 
	hipMalloc3DArray(&ga, &texf_desc, volumeSize,hipArrayLayered);
	texg.addressMode[0] = hipAddressModeWrap;
	texg.addressMode[1] = hipAddressModeWrap;
	texg.filterMode = hipFilterModeLinear;
	texg.normalized = true;
    hipBindTextureToArray(texg, ga,texf_desc);
    
    // // texture texg    
    // texf_desc = hipCreateChannelDesc<real>();	
    // volumeSize = make_hipExtent(n,nproj,nz);     
    // hipMalloc3DArray(&ga, &texf_desc, volumeSize,hipArrayLayered);
    
    // hipResourceDesc texgRes;
    // memset(&texgRes,0,sizeof(hipResourceDesc));
    // texgRes.resType            = hipResourceTypeArray;
    // texgRes.res.array.array    = ga;
    // hipTextureDesc             texgDescr;    
    // memset(&texgDescr,0,sizeof(hipTextureDesc));
    
	// texgDescr.addressMode[0] = hipAddressModeWrap;
	// texgDescr.addressMode[1] = hipAddressModeWrap;
	// texgDescr.filterMode = hipFilterModeLinear;
    // texgDescr.normalizedCoords = true;
    // texgDescr.readMode = hipReadModeNormalizedFloat;
    // hipArrayLayered
    // hipCreateTextureObject(&texg, &texgRes, &texgDescr, NULL);
    
    is_free = false;    
}

// destructor, memory deallocation
cfunc::~cfunc() { free(); }

void cfunc::free() {
    if (!is_free) {
        hipfftDestroy(plan_forward);
        hipfftDestroy(plan_inverse);    
        hipFree(fl);
        hipFree(flc);
        is_free = true;
    }
}

void cfunc::setgrids(size_t fz_, size_t lp2p1_, size_t lp2p2_, size_t lp2p1w_, size_t lp2p2w_, 
    size_t C2lp1_, size_t C2lp2_, size_t lpids_, size_t wids_, size_t cids_, 
    size_t nlpids_, size_t nwids_, size_t ncids_){
        
    fz = (complex*)fz_;
    lp2p1 = (float*)lp2p1_;
    lp2p2 = (float*)lp2p2_;
    lp2p1w = (float*)lp2p1w_;
    lp2p2w = (float*)lp2p2w_;
    C2lp1 = (float*)C2lp1_;
    C2lp2 = (float*)C2lp2_;
    lpids = (int*)lpids_;
    wids = (int*)wids_;
    cids = (int*)cids_;
    nlpids = nlpids_;
    nwids = nwids_;
    ncids = ncids_;        
}

void cfunc::backprojection(size_t f_, size_t g_, size_t stream_) 
{
    real* f = (real*)f_;
    real* g = (real*)g_;
    hipStream_t stream = (hipStream_t)stream_;
    // set thread block and grid sizes
    dim3 dimBlock(BS1,BS2,BS3);    
    uint GS1, GS2, GS3;    
    
    hipfftSetStream(plan_forward, stream);
    hipfftSetStream(plan_inverse, stream);    
    hipMemsetAsync(f, 0, nz*n*n*sizeof(real),stream); 

	//transpose for optimal cache usage
	GS1 = (uint)ceil(n/(float)BS1); GS2 = (uint)ceil(nproj/(float)BS2);GS3 = (uint)ceil(nz/(float)BS3);dim3 dimGrid1(GS1,GS2,GS3);    
	transpose<<<dimGrid1,dimBlock, 0, stream>>>(gtmp, g,n, nproj,nz);
	//compensate in samples for x direction
	GS1 = (uint)ceil(nproj/(float)BS1);GS2 = (uint)ceil(nz/(float)BS2); dim3 dimGrid2(GS1,GS2,1);    	
	SamplesToCoefficients2DY<<<dimGrid2, dimBlock, 0, stream>>>(gtmp,nproj*sizeof(real),nproj, n,nz);
	// //transpose back
	GS1 = (uint)ceil(nproj/(float)BS1);GS2 = (uint)ceil(n/(float)BS2);GS3 = (uint)ceil(nz/(float)BS3); dim3 dimGrid3(GS1,GS2,GS3);    	
	transpose<<<dimGrid3,dimBlock, 0, stream>>>(g,gtmp,nproj, n,nz);
	//compensate in samples for y direction
	GS1 = (uint)ceil(n/(float)BS1);GS2 = (uint)ceil(nz/(float)BS2); dim3 dimGrid4(GS1,GS2,1); 
	SamplesToCoefficients2DY<<<dimGrid4, dimBlock, 0, stream>>>(g,n*sizeof(real),n,nproj,nz);

    copy3DDeviceToArray(ga,g,make_hipExtent(n, nproj, nz),stream);

    //iterations over log-polar angular spans
    for(int k=0; k<3;k++)
    {
        hipMemsetAsync(fl, 0, nz*ntheta*nrho*sizeof(real),stream); 
		//interp from polar to log-polar grid
        GS1 = (uint)ceil(ceil(sqrt(nlpids))/(float)BS1); GS2 = (uint)ceil(ceil(sqrt(nlpids))/(float)BS2);GS3 = (uint)ceil(nz/(float)BS3);dim3 dimGrid1(GS1,GS2,GS3);    
        interp<<<dimGrid1, dimBlock, 0, stream>>>(0, fl,&lp2p2[k*nlpids],&lp2p1[k*nlpids],BS1*GS1,nlpids,n,nproj,nz,lpids,ntheta*nrho);
		//interp from polar to log-polar grid additional points
        GS1 = (uint)ceil(ceil(sqrt(nwids))/(float)BS1); GS2 = (uint)ceil(ceil(sqrt(nwids))/(float)BS2);GS3 = (uint)ceil(nz/(float)BS3);dim3 dimGrid2(GS1,GS2,GS3);    
        interp<<<dimGrid2, dimBlock, 0, stream>>>(2, fl,&lp2p2w[k*nwids],&lp2p1w[k*nwids],BS1*GS1,nwids,n,nproj,nz,wids,ntheta*nrho);
        //Forward FFT
        hipfftXtExec(plan_forward, fl,flc,HIPFFT_FORWARD);        
		//multiplication by adjoint fZ
        GS1 = (uint)ceil((ntheta/2+1)/(float)BS1); GS2 = (uint)ceil(nrho/(float)BS2);GS3 = (uint)ceil(nz/(float)BS3);dim3 dimGrid3(GS1,GS2,GS3);    
        mul<<<dimGrid3, dimBlock, 0, stream>>>(flc,fz,ntheta/2+1,nrho,nz);
		//Inverse FFT
        hipfftXtExec(plan_inverse,flc,fl,HIPFFT_BACKWARD);        
        //copy to binded texture 
        copy3DDeviceToArray(fla,fl,make_hipExtent(ntheta, nrho, nz),stream);
        //interp from log-polar to Cartesian grid
        GS1 = (uint)ceil(ceil(sqrt(ncids))/(float)BS1); GS2 = (uint)ceil(ceil(sqrt(ncids))/(float)BS2);GS3 = (uint)ceil(nz/(float)BS3);dim3 dimGrid4(GS1,GS2,GS3);
		interp<<<dimGrid4, dimBlock, 0, stream>>>(3, f,&C2lp1[k*ncids],&C2lp2[k*ncids],BS1*GS1,ncids,ntheta,nrho,nz,cids,n*n);                    
    }
}

