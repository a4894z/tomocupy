#include "hip/hip_runtime.h"
#include "cfunc.cuh"
#include "kernels.cuh"
#include <stdio.h>
cfunc::cfunc(int nproj, int nz, int n, int nrho, int ntheta):
nproj(nproj), nz(nz), n(n), nrho(nrho), ntheta(ntheta) {
    
    int ffts[] = {nrho,ntheta};
	int idist = nrho*ntheta;int odist = nrho*(ntheta/2+1);
	int inembed[] = {nrho, ntheta};int onembed[] = {nrho, ntheta/2+1};
	hipfftPlanMany(&plan_forward, 2, ffts, inembed, 1, idist, onembed, 1, odist, HIPFFT_R2C, nz); 
	hipfftPlanMany(&plan_inverse, 2, ffts, onembed, 1, odist, inembed, 1, idist, HIPFFT_C2R, nz); 

    hipMalloc((void **)&fl, nz*ntheta*nrho*sizeof(float)); 
	hipMalloc((void **)&flc, nz*(ntheta/2+1)*nrho*sizeof(float2)); 
    is_free = false;    
}

// destructor, memory deallocation
cfunc::~cfunc() { free(); }

void cfunc::setgrids(size_t fz_, size_t lp2p1_, size_t lp2p2_, size_t lp2p1w_, size_t lp2p2w_, 
    size_t C2lp1_, size_t C2lp2_, size_t lpids_, size_t wids_, size_t cids_, 
    size_t nlpids_, size_t nwids_, size_t ncids_){
        
    fz = (float2*)fz_;
    lp2p1 = (float*)lp2p1_;
    lp2p2 = (float*)lp2p2_;
    lp2p1w = (float*)lp2p1w_;
    lp2p2w = (float*)lp2p2w_;
    C2lp1 = (float*)C2lp1_;
    C2lp2 = (float*)C2lp2_;
    lpids = (int*)lpids_;
    wids = (int*)wids_;
    cids = (int*)cids_;
    nlpids = nlpids_;
    nwids = nwids_;
    ncids = ncids_;
}

void cfunc::free() {
    if (!is_free) {
        hipfftDestroy(plan_forward);
        hipfftDestroy(plan_inverse);    
        hipFree(fl);
        hipFree(flc);
        is_free = true;
    }
}

void cfunc::backprojection(size_t f_, size_t g_) 
{
    float* f = (float*)f_;
    float* g = (float*)g_;

    uint BS1 = 32; uint BS2 = 32; uint BS3 = 1;    
	uint GS1, GS2, GS3;    
    dim3 dimBlock(BS1,BS2,BS3);

    for(int k=0; k<3;k++)
    {
        hipMemset(fl, 0, nz*ntheta*nrho*sizeof(float)); 
		//interp from polar to log-polar grid

        GS1 = (uint)ceil(ceil(sqrt(nlpids))/(float)BS1); GS2 = (uint)ceil(ceil(sqrt(nlpids))/(float)BS2);GS3 = (uint)ceil(nz/(float)BS3);dim3 dimGrid1(GS1,GS2,GS3);
		interplp<<<dimGrid1, dimBlock>>>(fl,g,&lp2p2[k*nlpids],&lp2p1[k*nlpids],BS1*GS1,nlpids,n,nproj,nz,lpids,ntheta*nrho);
		//interp from polar to log-polar grid additional points
        GS1 = (uint)ceil(ceil(sqrt(nwids))/(float)BS1); GS2 = (uint)ceil(ceil(sqrt(nwids))/(float)BS2);GS3 = (uint)ceil(nz/(float)BS3);dim3 dimGrid2(GS1,GS2,GS3);
		interplp<<<dimGrid2, dimBlock>>>(fl,g,&lp2p2w[k*nwids],&lp2p1w[k*nwids],BS1*GS1,nwids,n,nproj,nz,wids,ntheta*nrho);

		//Forward FFT
		hipfftExecR2C(plan_forward, (hipfftReal*)fl,(hipfftComplex*)flc);

		//multiplication by adjoint fZ
		GS1 = (uint)ceil((ntheta/2+1)/(float)BS1); GS2 = (uint)ceil(nrho/(float)BS2);GS3 = (uint)ceil(nz/(float)BS3);dim3 dimGrid3(GS1,GS2,GS3);
		mul<<<dimGrid3, dimBlock>>>(1/(float)(ntheta*nrho),flc,fz,ntheta/2+1,nrho,nz);

		//Inverse FFT
		hipfftExecC2R(plan_inverse,(hipfftComplex*)flc,(hipfftReal*)fl);

		//interp from log-polar to Cartesian grid
		GS1 = (uint)ceil(ceil(sqrt(ncids))/(float)BS1); GS2 = (uint)ceil(ceil(sqrt(ncids))/(float)BS2);GS3 = (uint)ceil(nz/(float)BS3);dim3 dimGrid4(GS1,GS2,GS3);
		interpc<<<dimGrid4, dimBlock>>>(f,fl,&C2lp1[k*ncids],&C2lp2[k*ncids],BS1*GS1,ncids,ntheta,nrho,nz,cids,n*n);            
        
    }
}
