#include "hip/hip_runtime.h"
#include "cfunc_fourierrec.cuh"
#include "kernels_fourierrec.cuh"
#include "defs.cuh"
#include<stdio.h>
cfunc_fourierrec::cfunc_fourierrec(size_t ntheta, size_t pnz, size_t n, size_t theta_)
    : ntheta(ntheta), pnz(pnz), n(n) {
    float eps = 1e-2;
    mu = -log(eps) / (2 * n * n);
    m = ceil(2 * n * 1 / PI * sqrt(-mu * log(eps) + (mu * n) * (mu * n) / 4));    
    hipMalloc((void **)&fdee,
            (2 * n + 2 * m) * (2 * n + 2 * m) * pnz * sizeof(float2));
    hipMalloc((void **)&x, n * ntheta * sizeof(float));
    hipMalloc((void **)&y, n * ntheta * sizeof(float));
    
    long long ffts[] = {2*n,2*n};
	long long idist = (2 * n + 2 * m) * (2 * n + 2 * m);long long odist = (2 * n + 2 * m) * (2 * n + 2 * m);
    long long inembed[] = {2 * n + 2 * m, 2 * n + 2 * m};long long onembed[] = {2 * n + 2 * m, 2 * n + 2 * m};
    size_t workSize = 0;

    hipfftCreate(&plan2d);
    hipfftXtMakePlanMany(plan2d, 
        2, ffts, 
        inembed, 1, idist, CUDA_C, 
        onembed, 1, odist, CUDA_C, 
        pnz, &workSize, CUDA_C);    
    // fft 1d
    hipfftCreate(&plan1d);
    ffts[0] = n;
    idist = n;
    odist = n;
    inembed[0] = n;
    onembed[0] = n;
    hipfftXtMakePlanMany(plan1d, 
        1, ffts, 
        inembed, 1, idist, CUDA_C, 
        onembed, 1, odist, CUDA_C, 
        ntheta*pnz, &workSize, CUDA_C);                   

    theta = (float*)theta_;

  }


// destructor, memory deallocation
cfunc_fourierrec::~cfunc_fourierrec() { free(); }

void cfunc_fourierrec::free() {
  if (!is_free) {
    hipFree(fdee);
    hipFree(x);
    hipFree(y);
    hipfftDestroy(plan2d);
    hipfftDestroy(plan1d);
    is_free = true;   
  }
}

void cfunc_fourierrec::backprojection(size_t f_, size_t g_, size_t stream_) {
    float2* g = (float2 *)g_;    
    float2* f = (float2 *)f_;
    hipStream_t stream = (hipStream_t)stream_;    
    hipfftSetStream(plan1d, stream);
    hipfftSetStream(plan2d, stream);    

    // set thread block, grid sizes will be computed before cuda kernel execution
    dim3 dimBlock(BS1,BS2,BS3);    
    dim3 GS2d0,GS3d0,GS3d1,GS3d2,GS3d3;  
    GS2d0 = dim3(ceil(n / (float)BS1), ceil(ntheta / (float)BS2));
    GS3d0 = dim3(ceil(n / (float)BS1), ceil(n / (float)BS2),ceil(pnz / (float)BS3));
    GS3d1 = dim3(ceil(2 * n / (float)BS1), ceil(2 * n / (float)BS2),ceil(pnz / (float)BS3));
    GS3d2 = dim3(ceil((2 * n + 2 * m) / (float)BS1),ceil((2 * n + 2 * m) / (float)BS2), ceil(pnz / (float)BS3));
    GS3d3 = dim3(ceil(n / (float)BS1), ceil(ntheta / (float)BS2),ceil(pnz / (float)BS3));
   
    
    hipMemsetAsync(fdee, 0, (2 * n + 2 * m) * (2 * n + 2 * m) * pnz * sizeof(float2),stream);

    takexy <<<GS2d0, dimBlock, 0, stream>>> (x, y, theta, n, ntheta);

    ifftshiftc <<<GS3d3, dimBlock, 0, stream>>> (g, n, ntheta, pnz);
    hipfftXtExec(plan1d, (hipfftComplex *)g, (hipfftComplex *)g, HIPFFT_FORWARD);
    ifftshiftc <<<GS3d3, dimBlock, 0, stream>>> (g, n, ntheta, pnz);
    gather <<<GS3d3, dimBlock, 0, stream>>> (g, fdee, x, y, m, mu, n, ntheta, pnz, TOMO_ADJ);
    

    wrap <<<GS3d2, dimBlock, 0, stream>>> (fdee, n, pnz, m, TOMO_ADJ);

    fftshiftc <<<GS3d2, dimBlock, 0, stream>>> (fdee, 2 * n + 2 * m, pnz);
    hipfftXtExec(plan2d, (hipfftComplex *)&fdee[m + m * (2 * n + 2 * m)],
                (hipfftComplex *)&fdee[m + m * (2 * n + 2 * m)], HIPFFT_BACKWARD);
    fftshiftc <<<GS3d2, dimBlock, 0, stream>>> (fdee, 2 * n + 2 * m, pnz);
    
    divphi <<<GS3d0, dimBlock, 0, stream>>> (fdee, f, mu, n, pnz, m, TOMO_ADJ);    
}